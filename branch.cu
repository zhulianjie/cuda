#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void Warmup(float* c, int size)
{
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    float a = 0.0f;
    float b = a;

        a = 100.0f;
        b = 200.0F;

    if (tid < size)
        c[tid] = a+ b;
}

__global__ void k1(float* c, int size)
{
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    float a = 0.0f;
    float b = a;

    if (tid % 2 == 0)
        a = 100.0f;
    else
        b = 200.0F;

    if (tid < size)
        c[tid] = a+ b;
}

__global__ void k2(float* c, int size)
{
    int tid = threadIdx.x + blockIdx.x + blockDim.x;
    float a = 0.0f;
    float b = a;

   
    if (tid % 2 == 0)
        a = 102.0f;
    else if (tid % 3 == 0)
        a = 100.0f;
    else
        b = 200.0F;



    if (tid < size)
        c[tid] = a+ b;
        
        if (tid % 2 == 0)
        a = 102.0f;
    else if (tid % 3 == 0)
        a = 100.0f;
    else
        b = 200.0F;

    if (tid < size)
        c[tid] = a+ b;
}

template<class Func>
void RunAndPrintTime(char* funcName, Func&& func)
{
    using Clock = std::chrono::high_resolution_clock;
    auto now = Clock::now();
    func();
    auto end = Clock::now();
    std::cout << "total time " 
        << std::chrono::duration_cast<std::chrono::microseconds>(end - now).count()
         << " us" << std::endl;
}

dim3 block;
dim3 grid;
int size;

int main(int, char** argv)
{
    using Clock = std::chrono::high_resolution_clock;

    int blockSize = atoi(argv[1]);
    size = atoi(argv[2]);

    block.x = blockSize;
    grid.x = (size + block.x - 1)/ block.x;

    float* d_C = nullptr;
    hipMalloc(&d_C, sizeof(float) * size);

    hipDeviceSynchronize();

    // run warmup kernel
    RunAndPrintTime("Warmup", [d_C](){Warmup<<<grid, block>>>(d_C, size);});

    RunAndPrintTime("k1", [d_C](){k1<<<grid, block>>>(d_C, size);});

    RunAndPrintTime("k2", [d_C](){k2<<<grid, block>>>(d_C, size);});

}

